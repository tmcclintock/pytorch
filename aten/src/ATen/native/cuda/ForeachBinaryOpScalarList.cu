#include <ATen/Dispatch.h>
#include <ATen/native/ForeachUtils.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>

namespace at { namespace native {

template<template<class> class Op>
std::vector<Tensor> foreach_binary_op(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    std::vector<std::vector<at::Tensor>> tensor_lists;
    std::vector<at::Tensor> vec_res;
    vec_res.reserve(tensors.size());
    for (const auto& t: tensors) {
        vec_res.emplace_back(at::native::empty_like(t));
    }

    tensor_lists.emplace_back(tensors.vec());
    tensor_lists.emplace_back(vec_res);

    AT_DISPATCH_ALL_TYPES_AND3(kBFloat16, kHalf, kBool, tensors[0].scalar_type(), "foreach_binary_op_scalarlist_cuda", [&]() {
        using opmath_t = get_opmath_t<scalar_t>::opmath_t;
        multi_tensor_apply<2, opmath_t>(tensor_lists,
                                        scalars,
                                        BinaryOpScalarListFunctor<scalar_t,
                                                                  /* depth */ 2,
                                                                  /* r_args_depth */ 1,
                                                                  /* res_arg_index */ 1>(),
                                        Op<opmath_t>());
    });
    return tensor_lists[1];
}

template<template<class> class Op>
void foreach_binary_op_(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    std::vector<std::vector<at::Tensor>> tensor_lists;
    tensor_lists.emplace_back(tensors.vec());

    AT_DISPATCH_ALL_TYPES_AND3(kBFloat16, kHalf, kBool, tensors[0].scalar_type(), "foreach_binary_op_scalarlist_cuda_", [&]() {
        using opmath_t = get_opmath_t<scalar_t>::opmath_t;
        multi_tensor_apply<1, opmath_t>(tensor_lists,
                                        scalars,
                                        BinaryOpScalarListFunctor<scalar_t,
                                                                    /* depth */ 1,
                                                                    /* r_args_depth */ 1,
                                                                    /* res_arg_index */ 0>(),
                                        Op<opmath_t>());
    });
}

#define FOREACH_BINARY_OP_SCALARLIST(NAME, OP)                                                                          \
void foreach_tensor_##NAME##_scalarlist_kernel_cuda_(TensorList tensors, at::ArrayRef<Scalar> scalars) {                \
    check_foreach_api_restrictions(tensors, scalars);                                                                   \
    if (!can_use_fast_route(tensors, scalars)) {                                                                        \
        return at::native::foreach_tensor_##NAME##_scalarlist_kernel_slow_(tensors, scalars);                           \
    }                                                                                                                   \
                                                                                                                        \
    foreach_binary_op_<OP>(tensors, scalars);                                                                           \
}                                                                                                                       \
                                                                                                                        \
std::vector<Tensor> foreach_tensor_##NAME##_scalarlist_kernel_cuda(TensorList tensors, at::ArrayRef<Scalar> scalars) {  \
    check_foreach_api_restrictions(tensors, scalars);                                                                   \
    if (!can_use_fast_route(tensors, scalars)) {                                                                        \
        return at::native::foreach_tensor_##NAME##_scalarlist_kernel_slow(tensors, scalars);                            \
    }                                                                                                                   \
                                                                                                                        \
    return foreach_binary_op<OP>(tensors, scalars);                                                                     \
}

FOREACH_BINARY_OP_SCALARLIST(add, std::plus);
FOREACH_BINARY_OP_SCALARLIST(mul, std::multiplies);

// In the case of division, integer inputs will result in float. 
// Currently multi tensor apply can only return result of the same type as input.
void foreach_tensor_div_scalarlist_kernel_cuda_(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    check_foreach_api_restrictions(tensors, scalars);
    if (!can_use_fast_route(tensors, scalars, /*div_op*/ true)) {
        return at::native::foreach_tensor_div_scalarlist_kernel_slow_(tensors, scalars);
    }

    foreach_binary_op_<std::divides>(tensors, scalars);
}

std::vector<Tensor> foreach_tensor_div_scalarlist_kernel_cuda(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    check_foreach_api_restrictions(tensors, scalars);
    if (!can_use_fast_route(tensors, scalars, /*div_op*/ true)) {
        return at::native::foreach_tensor_div_scalarlist_kernel_slow(tensors, scalars);
    }

    return foreach_binary_op<std::divides>(tensors, scalars);
}

// In the case of subtraction, we dont allow scalar to be boolean following the torch.sub logic
void foreach_tensor_sub_scalarlist_kernel_cuda_(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    check_foreach_api_restrictions(tensors, scalars);

    TORCH_CHECK(tensors[0].scalar_type() != kBool || !scalars[0].isBoolean(),
              "Subtraction, the `-` operator, with two bool tensors is not supported. "
              "Use the `^` or `logical_xor()` operator instead.")
    TORCH_CHECK(tensors[0].scalar_type() != kBool && !scalars[0].isBoolean(),
              "Subtraction, the `-` operator, with a bool tensor is not supported. "
              "If you are trying to invert a mask, use the `~` or `logical_not()` operator instead.");

    if (!can_use_fast_route(tensors, scalars)) {
        return at::native::foreach_tensor_sub_scalarlist_kernel_slow_(tensors, scalars);
    }

    foreach_binary_op_<std::minus>(tensors, scalars);
}

std::vector<Tensor> foreach_tensor_sub_scalarlist_kernel_cuda(TensorList tensors, at::ArrayRef<Scalar> scalars) {
    check_foreach_api_restrictions(tensors, scalars);

    TORCH_CHECK(tensors[0].scalar_type() != kBool || !scalars[0].isBoolean(),
              "Subtraction, the `-` operator, with two bool tensors is not supported. "
              "Use the `^` or `logical_xor()` operator instead.")
    TORCH_CHECK(tensors[0].scalar_type() != kBool && !scalars[0].isBoolean(),
              "Subtraction, the `-` operator, with a bool tensor is not supported. "
              "If you are trying to invert a mask, use the `~` or `logical_not()` operator instead.");

    if (!can_use_fast_route(tensors, scalars)) {
        return at::native::foreach_tensor_sub_scalarlist_kernel_slow(tensors, scalars);
    }

    return foreach_binary_op<std::minus>(tensors, scalars);
}

}} // namespace at::native
